#include "utility.cuh"

void __cudaSafeCall(hipError_t error, const char *file, const int line){
	if(hipSuccess != error)
		printf("Error in %s, line %i: %s\n", file, line, hipGetErrorString(error));
}

void __cudaErrorChecker(const char *file, const int line){
	hipError_t error = hipGetLastError();
	if(hipSuccess != error)
		printf("Error in %s, line %i: %s\n", file, line, hipGetErrorString(error));
}

Mouse getMouse(Scene const &scene){
	//if click on card, flip it
	double mouseX, mouseY;
	glfwGetCursorPos(scene.window, &mouseX, &mouseY);
	mouseX = mouseX/scene.width * 2 - 1.0;
	mouseY = -mouseY/scene.height * 2 + 1.0; //mouseY is bottom down
	vec2 mousePos = {static_cast<float>(mouseX), static_cast<float>(mouseY)};

	bool pressed = glfwGetMouseButton(scene.window, GLFW_MOUSE_BUTTON_LEFT);

	return {mousePos, pressed};
}