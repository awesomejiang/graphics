#include "utility.cuh"

void __cudaSafeCall(hipError_t error, const char *file, const int line){
	if(hipSuccess != error)
		printf("Error in %s, line %i: %s\n", file, line, hipGetErrorString(error));
}

void __cudaErrorChecker(const char *file, const int line){
	hipError_t error = hipGetLastError();
	if(hipSuccess != error)
		printf("Error in %s, line %i: %s\n", file, line, hipGetErrorString(error));
}


__DEVICE__ int getIdx(){
	int grid = gridDim.x*gridDim.y*blockIdx.z + gridDim.x*blockIdx.y + blockIdx.x;
	return blockDim.x*grid + threadIdx.x;
}


Mouse getMouse(Scene const &scene){
	//if click on card, flip it
	double mouseX, mouseY;
	glfwGetCursorPos(scene.window, &mouseX, &mouseY);
	mouseX = mouseX/scene.width * 2 - 1.0;
	mouseY = -mouseY/scene.height * 2 + 1.0; //mouseY is bottom down
	vec2 mousePos = {static_cast<float>(mouseX), static_cast<float>(mouseY)};

	bool pressed = glfwGetMouseButton(scene.window, GLFW_MOUSE_BUTTON_LEFT);

	return {mousePos, pressed};
}