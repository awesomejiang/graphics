#include "hip/hip_runtime.h"
#include "spirit.cuh"

#define MAX_THREAD 1024
#define MAX_BLOCK_X 65535ll
#define MAX_BLOCK_Y 65535ll
#define MAX_BLOCK_Z 65535ll

Spirit::Spirit(std::vector<Particle> particles)
: particles(particles),
  resource(0),
  nParticle(particles.size()),
  pShader("shaders/particle.vs", "shaders/particle.fs"){
	createVBO();
	setCallBacks();
	initCuda();
}

Spirit::~Spirit(){
	//unmap resource
	CUDA_SAFE_CALL( hipGraphicsUnmapResources(1, &resource) );
	CUDA_SAFE_CALL( hipGraphicsUnregisterResource(resource) );
}

void Spirit::createVBO(){
	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);

	glBindVertexArray(VAO);

	//set VBO
	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, particles.size()*sizeof(Particle), particles.data(), GL_STATIC_DRAW);

	//set VAO
	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(0));
	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(sizeof(vec2)*1));
	glEnableVertexAttribArray(2);
	glVertexAttribPointer(2, 4, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)(sizeof(vec2)*2));

	//unbind
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

}

void Spirit::setCallBacks() const{
	//glfwSetCursorPosCallback(scene.window, [](GLFWWindow *window, float x, float y){});
}


void Spirit::initCuda(){
	deployGrid();

	//cuda allocations
	auto sz = nParticle*sizeof(Particle);
	Particle* deviceParticles = nullptr;
	CUDA_SAFE_CALL( hipMalloc((void**)&deviceParticles, sz) );
	CUDA_SAFE_CALL( hipMemcpy(deviceParticles, particles.data(), sz, hipMemcpyHostToDevice) );

	//register to cuda
	CUDA_SAFE_CALL( hipGraphicsGLRegisterBuffer(&resource, VBO, hipGraphicsRegisterFlagsNone) );

	//map dptr to VBO
	size_t retSz;
	Particle *dptr = nullptr;
	CUDA_SAFE_CALL( hipGraphicsMapResources(1, &resource) );
	CUDA_SAFE_CALL( hipGraphicsResourceGetMappedPointer((void**)&dptr, &retSz, resource) );

	//run cuda kernel
	initKernel<<<grid, block>>>(dptr, nParticle, deviceParticles);
	CUDA_ERROR_CHECKER;

	//free
	CUDA_SAFE_CALL( hipFree(deviceParticles) );
}


void Spirit::render(Scene const &scene){
	double mouseX, mouseY;
	glfwGetCursorPos(scene.window, &mouseX, &mouseY);
	mouseX = mouseX/scene.width * 2 - 1.0;
	mouseY = -mouseY/scene.height * 2 + 1.0; //mouseY is bottom down
	vec2 mousePos = {static_cast<float>(mouseX), static_cast<float>(mouseY)};

	int state = glfwGetMouseButton(scene.window, GLFW_MOUSE_BUTTON_LEFT);

	//get mouse position
	vec2* deviceMousePos = nullptr;
	auto sz = sizeof(vec2);
	CUDA_SAFE_CALL( hipMalloc((void**)&deviceMousePos, sz) );
	CUDA_SAFE_CALL( hipMemcpy(deviceMousePos, &mousePos, sz, hipMemcpyHostToDevice) );

	//map dptr to VBO
	size_t retSz;
	Particle *dptr = nullptr;
	CUDA_SAFE_CALL( hipGraphicsResourceGetMappedPointer((void**)&dptr, &retSz, resource) );
	//run cuda kernel
	renderKernel<<<block, grid>>>(dptr, nParticle, deviceMousePos, state);
	CUDA_ERROR_CHECKER;

	//draw
	pShader.use();

    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE);

	glBindVertexArray(VAO);
	glDrawArrays(GL_POINTS, 0, particles.size());
	glBindVertexArray(0);

	glDisable(GL_BLEND);

	//free
	CUDA_SAFE_CALL( hipFree(deviceMousePos) );
}

__global__ void initKernel(Particle* dptr, int n, Particle *p){
    int index = getIdx();
    if(index > n)
    	return ;

    dptr[index] = p[index];
}

__global__ void renderKernel(Particle* dptr, int n, vec2 *pos, int state){
    int index = getIdx();
    if(index > n)
    	return ;

    bool pressed = state==GLFW_PRESS? true: false;
    dptr[index].update(*pos, pressed);
}

__device__ int getIdx(){
	int grid = gridDim.x*gridDim.y*blockIdx.z + gridDim.x*blockIdx.y + blockIdx.x;
	return blockDim.x*grid + threadIdx.x;
}


void Spirit::deployGrid(){
	unsigned int blockX = nParticle>MAX_THREAD? MAX_THREAD: static_cast<unsigned int>(nParticle);
	block = {blockX, 1, 1};

	float nGrid = static_cast<float>(nParticle)/blockX;
	if(nGrid > MAX_BLOCK_X*MAX_BLOCK_Y*MAX_BLOCK_Z)
		throw std::runtime_error("Number of particles out of gpu limits.");
	else if(nGrid > MAX_BLOCK_X*MAX_BLOCK_Y){
		unsigned int z = std::ceil(nGrid/MAX_BLOCK_X/MAX_BLOCK_Y);
		grid = {MAX_BLOCK_X, MAX_BLOCK_Y, z};
	}
	else if(nGrid > MAX_BLOCK_X){
		unsigned int y = std::ceil(nGrid/MAX_BLOCK_X);
		grid = {MAX_BLOCK_X, y, 1};
	}
	else if(nGrid > 0){
		unsigned int x = std::ceil(nGrid);
		grid = {x, 1, 1};
	}
	else
		throw std::runtime_error("No particles in screen.");
}